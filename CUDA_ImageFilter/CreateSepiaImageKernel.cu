#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <assert.h>

#include "ImagePixel.h"

#define min(a, b) (a > b)? b: a
#define max(a, b) (a > b)? a: b 

__global__ void CreateSepiaImageKernel(struct ImagePixel* inputPixels, struct ImagePixel* outputPixels, unsigned int width, unsigned int height)
{
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	unsigned int index = width * y + x;

	if (x > width - 1 || y > height - 1) {
		// We are out of image size
		return;
	}

	ImagePixel* inputPixel = &inputPixels[index];
	ImagePixel* outputPixel = &outputPixels[index];

	// BGR order is used
	outputPixel->B = min((unsigned int)((inputPixel->B * 0.393f) + (inputPixel->G * 0.769f) + (inputPixel->R * 0.189f)), 255U);
	outputPixel->G = min((unsigned int)((inputPixel->B * 0.349f) + (inputPixel->G * 0.686f) + (inputPixel->R * 0.168f)), 255U);
	outputPixel->R = min((unsigned int)((inputPixel->B * 0.272f) + (inputPixel->G * 0.534f) + (inputPixel->R * 0.131f)), 255U);
}

void MallocAndCopyPixelsToDevice(ImagePixel* pixelsHost, ImagePixel** pixelsDevice, unsigned int pixelsBytes)
{
	assert(hipMalloc((void **)pixelsDevice, pixelsBytes) == hipSuccess);
	assert(hipMemcpy(*pixelsDevice, pixelsHost, pixelsBytes, hipMemcpyHostToDevice) == hipSuccess);
}

void CreateSepiaImageOnGPU(ImagePixel* inputPixels, ImagePixel* outputPixels, unsigned int width, unsigned int height)
{
	int pixelsCount = width * height;
	int pixelsBytes = sizeof(ImagePixel) * pixelsCount;

	// Output pixels on device - malloc
	ImagePixel* outputPixelsDevice;
	MallocAndCopyPixelsToDevice(outputPixels, &outputPixelsDevice, pixelsBytes);

	// Input pixels on device - malloc and copy to VRAM
	ImagePixel* inputPixelsDevice;
	MallocAndCopyPixelsToDevice(inputPixels, &inputPixelsDevice, pixelsBytes);

	dim3 threads(32, 32);
	dim3 blocks(width / threads.x, height / threads.y);

	CreateSepiaImageKernel<<<blocks, threads>>>(inputPixelsDevice, outputPixelsDevice, width, height);
	assert(hipPeekAtLastError() == hipSuccess);
	hipDeviceSynchronize();

	// Copy from device to CPU memory pointer
	assert(hipMemcpy(outputPixels, outputPixelsDevice, pixelsBytes, hipMemcpyDeviceToHost) == hipSuccess);

	// Free device memory
	hipFree(outputPixelsDevice);
	hipFree(inputPixelsDevice);

	assert(hipPeekAtLastError() == hipSuccess);

	hipDeviceReset();
}

#undef min
#undef max