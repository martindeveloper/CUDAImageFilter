#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <assert.h>

#include "ImagePixel.h"

__global__ void CreateGrayscaleImageKernel(struct ImagePixel* inputPixels, struct ImagePixel* outputPixels, unsigned int width, unsigned int height)
{
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	unsigned int index = width * y + x;

	if (x > width - 1 || y > height - 1) {
		// We are out of image size
		return;
	}

	ImagePixel* inputPixel = &inputPixels[index];
	ImagePixel* outputPixel = &outputPixels[index];

	unsigned char gray = inputPixel->B * 0.299f + inputPixel->G * 0.587f + inputPixel->R * 0.114f;

	// BGR order is used
	outputPixel->B = gray;
	outputPixel->G = gray;
	outputPixel->R = gray;
}

void CreateGrayscaleImageOnGPU(ImagePixel* inputPixels, ImagePixel* outputPixels, unsigned int width, unsigned int height)
{
	int pixelsCount = width * height;
	int pixelsBytes = sizeof(ImagePixel) * pixelsCount;

	// Output pixels on device - malloc
	ImagePixel* outputPixelsDevice;
	assert(hipMalloc((void **)&outputPixelsDevice, pixelsBytes) == hipSuccess);
	assert(hipMemcpy(outputPixelsDevice, outputPixels, pixelsBytes, hipMemcpyHostToDevice) == hipSuccess);
	
	// Input pixels on device - malloc and copy to VRAM
	ImagePixel* inputPixelsDevice;
	assert(hipMalloc((void **)&inputPixelsDevice, pixelsBytes) == hipSuccess);
	assert(hipMemcpy(inputPixelsDevice, inputPixels, pixelsBytes, hipMemcpyHostToDevice) == hipSuccess);

	dim3 threads(32, 32);
	dim3 blocks(width / threads.x, height / threads.y);

	CreateGrayscaleImageKernel<<<blocks, threads>>>(inputPixelsDevice, outputPixelsDevice, width, height);
	assert(hipPeekAtLastError() == hipSuccess);

	hipDeviceSynchronize();

	// Copy from device to CPU memory pointer
	assert(hipMemcpy(outputPixels, outputPixelsDevice, pixelsBytes, hipMemcpyDeviceToHost) == hipSuccess);

	// Free device memory
	hipFree(outputPixelsDevice);
	hipFree(inputPixelsDevice);

	assert(hipPeekAtLastError() == hipSuccess);

	hipDeviceReset();
}