#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <assert.h>

#include "ImagePixel.h"

__global__ void CreateSepiaImageKernel(struct ImagePixel* inputPixels, struct ImagePixel* outputPixels, unsigned int width, unsigned int height)
{
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	unsigned int index = width * y + x;

	if (x > width - 1 || y > height - 1) {
		// We are out of image size
		return;
	}

	ImagePixel* inputPixel = &inputPixels[index];
	ImagePixel* outputPixel = &outputPixels[index];

	// BGR order is used
	outputPixel->B = (inputPixel->B * 0.393f) + (inputPixel->G * 0.769f) + (inputPixel->R * 0.189f);
	outputPixel->G = (inputPixel->B * 0.349f) + (inputPixel->G * 0.686f) + (inputPixel->R * 0.168f); 
	outputPixel->R = (inputPixel->B * 0.272f) + (inputPixel->G * 0.534f) + (inputPixel->R * 0.131f);
}

void CreateSepiaImageOnGPU(ImagePixel* inputPixels, ImagePixel* outputPixels, unsigned int width, unsigned int height)
{
	int pixelsCount = width * height;
	int pixelsBytes = sizeof(ImagePixel) * pixelsCount;

	// Output pixels on device - malloc
	ImagePixel* outputPixelsDevice;
	assert(hipMalloc((void **)&outputPixelsDevice, pixelsBytes) == hipSuccess);
	assert(hipMemcpy(outputPixelsDevice, outputPixels, pixelsBytes, hipMemcpyHostToDevice) == hipSuccess);

	// Input pixels on device - malloc and copy to VRAM
	ImagePixel* inputPixelsDevice;
	assert(hipMalloc((void **)&inputPixelsDevice, pixelsBytes) == hipSuccess);
	assert(hipMemcpy(inputPixelsDevice, inputPixels, pixelsBytes, hipMemcpyHostToDevice) == hipSuccess);

	dim3 threads(32, 32);
	dim3 blocks(width / threads.x, height / threads.y);

	CreateSepiaImageKernel<<<blocks, threads>>>(inputPixelsDevice, outputPixelsDevice, width, height);
	assert(hipPeekAtLastError() == hipSuccess);
	hipDeviceSynchronize();

	// Copy from device to CPU memory pointer
	assert(hipMemcpy(outputPixels, outputPixelsDevice, pixelsBytes, hipMemcpyDeviceToHost) == hipSuccess);

	// Free device memory
	hipFree(outputPixelsDevice);
	hipFree(inputPixelsDevice);

	assert(hipPeekAtLastError() == hipSuccess);

	hipDeviceReset();
}