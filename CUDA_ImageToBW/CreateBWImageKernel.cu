#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <assert.h>
#include "ImagePixel.h"

__global__ void CreateBWImageKernel(ImagePixel* inputPixels, ImagePixel* outputPixels)
{
	int pixelIndex = /*blockIdx.x * */threadIdx.x;

	ImagePixel* inputPixel = &inputPixels[pixelIndex];
	ImagePixel* outputPixel = &outputPixels[pixelIndex];

	// Data loss of fractional part
	outputPixel->R = inputPixel->R * 0.299f;
	outputPixel->G = inputPixel->G * 0.587f;
	outputPixel->B = inputPixel->B * 0.114f;
}

void CreateBWImage(ImagePixel* inputPixels, ImagePixel* outputPixels, int pixelsCount)
{
	int pixelsBytes = sizeof(ImagePixel) * pixelsCount;

	// Output pixels on device - malloc
	ImagePixel* outputPixelsDevice;
	assert(hipMalloc((void **)&outputPixelsDevice, pixelsBytes) == hipSuccess);

	// Input pixels on device - malloc and copy to VRAM
	ImagePixel* inputPixelsDevice;
	assert(hipMalloc((void **)&inputPixelsDevice, pixelsBytes) == hipSuccess);
	assert(hipMemcpy(inputPixelsDevice, inputPixels, pixelsBytes, hipMemcpyHostToDevice) == hipSuccess);

	// One block and pixelsCount threads
	dim3 blocks(0, 0, 0);
	dim3 threads(0, 0, 0);

	CreateBWImageKernel<<<1, 1>>>(inputPixelsDevice, outputPixelsDevice);
	hipDeviceSynchronize();

	// Copy from device to CPU memory pointer
	assert(hipMemcpy(outputPixels, outputPixelsDevice, pixelsBytes, hipMemcpyDeviceToHost) == hipSuccess);

	// Free device memory
	hipFree(outputPixelsDevice);
	hipFree(inputPixelsDevice);

	const char* error = hipGetErrorString(hipGetLastError());

	hipDeviceReset();
}